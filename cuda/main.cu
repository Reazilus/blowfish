// Written by Andrew Carter (2008)
// Modified by Tuan Dao (2016)

#include "blowfish.h"
#include "implement.h"
#include "const.h"
#include <stdio.h>
#include <string.h>
#include <stdint.h>

int main(int argc, char *argv[])
{
	splashscreen();
	version();
	// Misc variables
	int status = 0;
	uint64_t hash_original, hash_encrypted, hash_decrypted;
	float cudaRuntime, rate;
	// File variables
	size_t filesize;
	const char *filepath = "../testfile";
	uint32_t *file = (uint32_t*) readfile(&filesize, filepath);
	size_t numblocks = filesize/sizeof(uint32_t);
	printf("File size = %zu bytes, numblocks = %zu\n", filesize, numblocks/2);
	// Encryption key
	const char *key = "TESTKEY";
	printf("Key = %s, length = %zu\n", key, strlen(key));
	// Create Blowfish context
	blowfish_context_t *context = (blowfish_context_t*) malloc(sizeof(blowfish_context_t));
	if(!context) 
	{
		printf("Could not allocate enough memory!\n");
		return -1;
	}

	// Initialize key schedule
	status = blowfish_init(context, key, strlen(key));
	if (status)
	{
		printf("Error initiating key\n");
		return -1;
	} else printf("Key schedule complete!\n");

	// Hash original file
	hash_original = hash(file, numblocks);
	printf("Original hash = %llx\n", (unsigned long long)hash_original);

	// CUDA Starts
	printf("CUDA Starts!\n");

	uint32_t *filegpu;
	hipMalloc(&filegpu, filesize);
	hipMemcpy(filegpu, file, filesize, hipMemcpyHostToDevice);

	blowfish_context_t *ctxgpu;
	hipMalloc(&ctxgpu, sizeof(blowfish_context_t));
	hipMemcpy(ctxgpu, context, sizeof(blowfish_context_t), hipMemcpyHostToDevice);

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);

	int blocksize = 1024;
	int gridsize = (numblocks/(blocksize<<1)+1);

	//__________ENCRYPTION__________
	printf("Encryption starts...\n");

	hipEventRecord(start);
	cudaBlowfishEncryptPtr<<<gridsize, blocksize>>>(ctxgpu, filegpu, numblocks);
	hipEventRecord(stop);

	hipMemcpy(file, filegpu, filesize, hipMemcpyDeviceToHost);	
	hipEventSynchronize(stop);

	hash_encrypted = hash(file, numblocks);

	hipEventElapsedTime(&cudaRuntime, start, stop);
	rate = (filesize*1e3)/(cudaRuntime);

	printf("Encryption done!\n");
	printf("Time taken: %lf milliseconds\n", cudaRuntime);
	printf("Average speed: %lf MB/s\n", rate/MEGABYTE);
	printf("Encrypted hash = %llx\n", (unsigned long long)hash_encrypted);

	//__________DECRYPTION__________
	printf("Encryption starts...\n");

	hipEventRecord(start);
	cudaBlowfishDecryptPtr<<<gridsize, blocksize>>>(ctxgpu, filegpu, numblocks);
	hipEventRecord(stop);

	hipMemcpy(file, filegpu, filesize, hipMemcpyDeviceToHost);	
	hipEventSynchronize(stop);

	hash_decrypted = hash(file, numblocks);

	hipEventElapsedTime(&cudaRuntime, start, stop);
	rate = (filesize*1e3)/(cudaRuntime);

	printf("Decryption done!\n");
	printf("Time taken: %lf milliseconds\n", cudaRuntime);
	printf("Average speed: %lf MB/s\n", rate/MEGABYTE);
	printf("Decrypted hash = %llx\n", (unsigned long long)hash_decrypted);

	// Check
	if (hash_decrypted == hash_original)
		printf("Hashes match! PASSED!\n");
	else
		printf("Hashes mismatch! FAILED!\n");

	//__________DONE__________
	blowfish_clean(context);
	free(file);
	hipFree(filegpu);
	hipFree(ctxgpu);
	return 0;
}